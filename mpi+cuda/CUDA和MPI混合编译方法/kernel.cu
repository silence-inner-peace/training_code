#include "hip/hip_runtime.h"

#include "cuda.cuh"

__global__ void matadd_kernel(float *matA, float *matB, float *matC, int nx, int ny, int nz)
{
   int ix = blockDim.x * blockIdx.x + threadIdx.x;
   int iy = blockDim.y * blockIdx.y + threadIdx.y;

   for (int iz = 0; iz < nz; iz ++)
   {
     if (ix < nx && iy < ny)
        matC[iz * ny * nx + iy * nx + ix] = matA[iz * ny * nx + iy * nx + ix] + matB[iz * ny * nx + iy * nx + ix];
   }
}

 void GPU_add(float *d_loc_matA,float * d_loc_matB,float * d_loc_matC,int nx,int ny,int loc_nz,dim3 dimBlock,dim3 dimGrid)
 {
	   matadd_kernel<<<dimGrid, dimBlock>>>(d_loc_matA, d_loc_matB, d_loc_matC, nx, ny, loc_nz);
 }


