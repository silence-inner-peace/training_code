#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>


#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

__device__  volatile  int  vint = 0;
//a#########################
__global__ void fun ( float  * vp_device , int n, int nz, int  is )
{
        int it = threadIdx.x + blockDim.x * blockIdx.x;
        if  ( it < n ) {
                vp_device[it]=2000;
                if ( ( it > nz * 40 && it < 40 && it % nz < 60 ) ) 
                        vp_device [ it ] = 2500 * is * 100 ;
        }
}
//a########################
extern "C" void cudaFun ( int is , FILE  *fp ,  int  nx , int nz )
{
        int i ;
        float  * vp_device , * vp_host;

        hipMalloc(&vp_device, nx*nz*sizeof(float));  
        hipMemset(vp_device, 0, nx*nz*sizeof(float));

        vp_host=(float*)malloc(nx*nz*sizeof(float));

        float mstimer;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        fun <<<(  nx * nz   +   511 ) / 512, 512>>> ( vp_device ,  nx*nz , nz , is ) ;

        hipMemcpy(vp_host, vp_device, nx*nz*sizeof(float),hipMemcpyDeviceToHost); 

        fseek(fp,is*nx*nz*sizeof(float),0);
        for (  i  =  0  ;  i  <  nx  *  nz   ;  i   ++   )
                fwrite( &vp_host[i] , sizeof(float) , 1 , fp);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&mstimer, start, stop);
        printf( "CUDA : is = %d, time = %g (s)\\n " ,is, mstimer/1000);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        hipFree(vp_device);
        free(vp_host);
}
