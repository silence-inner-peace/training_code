#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "vector.h"
#include "vector_kernel.h"

void vectorAdd(float *a, float *b, float *c, int n)
{
	float *d_a, *d_b, *d_c;
	hipMalloc((void **)&d_a, n * sizeof(float));
	hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_b, n * sizeof(float));
	hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_c, n * sizeof(float));
	hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);
	vectorAddKernel<<<1, n>>>(d_a, d_b, d_c, n);
	hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}
