#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "vector_kernel.h"

__global__ void vectorAddKernel(float *a, float *b, float *c, int n)
{
	int tid;
	tid = threadIdx.x;
	if(tid < n)
		c[tid] = a[tid] + b[tid];
}
