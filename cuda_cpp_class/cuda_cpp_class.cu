#include <stdio.h>

#include <iostream>

#include <hip/hip_runtime.h>



class MyClass;

__global__ void kernel(int *a, unsigned int N);



class MyClass {

public:

  MyClass(int len) {

    length = len;

    hipMalloc((void **)&d_data, sizeof(int)*length);

    hipMemset((void *)d_data, 0, sizeof(int)*length);

  };

  

  ~MyClass() {

    hipFree((void *)d_data);

    printf("%s\n","cudafree" );

  };

  

  void run(dim3 grid,dim3 block) {

    kernel<<<grid, block>>>(d_data, length);

  };





  void set(int* h_data)

  {

  hipMemcpy(d_data,h_data,sizeof(int)*length,hipMemcpyHostToDevice);

  }

  

  int* getData(void) {

    return d_data;

  };

  int getLength(void)

  {

    return length;

  }

  void show(void)

  {

    int h_data[length];

    hipMemcpy(h_data, getData(), sizeof(int)*length, hipMemcpyDeviceToHost);

    for (int i=0; i<length; i++) {

      std::cout << h_data[i] << " ";

    }

    std::cout << std::endl;

  }

public:

  int *d_data;

  int length;

};





__global__ void kernel(int *a, unsigned int N) 
{
  const unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i<N) {
    a[i] += i;
  }
}
__global__ void kernel1(int *a, unsigned int N) 
{
  const unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i<N) {
    a[i] += 2*i;
  }
}
class MyClass1:public MyClass
{
public:
  MyClass1(int len):MyClass(len){};
  void run(dim3 grid,dim3 block) 
  {
    kernel1<<<grid, block>>>(d_data, length);
  };
};

int main(void) {

  int arraySize = 20;
  int* testArr = new int[arraySize];
  for (int i = 0; i < arraySize; ++i)
  {
      testArr[i] = i;
  }

  // MyClass c(arraySize);  //直接声明的对象是定义在栈上的，会被自动释放  
  // c.run();
  // c.show();



  dim3 grid(1);

  dim3 block(arraySize);

  MyClass1 *c = new MyClass1(arraySize);

  c->set(testArr);

  c->run(grid,block);

  c->show();

  delete c; //用指针指向new出来的对象是存放在堆上的，必须要手动delete对象，否则对象不会被释放掉。

}
