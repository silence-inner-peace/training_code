#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "Array2D_CUDA.h"
using namespace std;

__global__ void kernel1(int* d_output, int *a, unsigned int N) 
{
  const unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i<N) {
    d_output[i] = a[i] + i;
  }
}
__global__ void kernel2(int* d_output, int *a, unsigned int N) 
{
  const unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i<N) {
    d_output[i] = a[i] + 2*i;
  }
}

class AbstractMetricsProcess
{
public:
	AbstractMetricsProcess(int rows, int cols)
	{
		int arraySize = rows*cols;
		int* testArr = new int[arraySize];
		for (int i = 0; i < arraySize; ++i)
		{
		  testArr[i] = i;
		}
		img = new Array2D< Cutype<int> >(rows,cols);
		img->set(testArr);
	}
	// void runCCL();
	virtual void calMetrics(dim3 grid,dim3 block) = 0;	//具体的指数计算类推迟到子类中实现
	void templateMethod(int rows, int cols, dim3 grid, dim3 block)
	{
    img->show();
    calMetrics(grid,block);
	}
  ~AbstractMetricsProcess()
  {
    delete img;
    img = NULL;
  }
public:
	Array2D< Cutype<int> >* img;
};



class ConcreateArea:public AbstractMetricsProcess 	
{
public:
	Array2D< Cutype<int> >* d_data1;  //每个派生类中的run函数封装一种对标记数组的操作，并将结果存储到派生类的成员变量中
public:
  ConcreateArea(int rows, int cols):AbstractMetricsProcess(rows,cols)
  {
    d_data1 = new Array2D< Cutype<int> >(rows,cols);
  };
  void calMetrics(dim3 grid,dim3 block) 
  {
    kernel1<<<grid, block>>>(d_data1->getDevData(), img->getDevData(), img->size());		//d_data来源于父类，d_data1来源于派生类
    d_data1->show();
  };
  ~ConcreateArea()
  {
    delete d_data1;
    d_data1 = NULL;
  }
};


class ConcreatePeri:public AbstractMetricsProcess
{
public:
	Array2D< Cutype<int> >* d_data2;
public:
  ConcreatePeri(int rows, int cols):AbstractMetricsProcess(rows,cols)
  {
    d_data2 = new Array2D< Cutype<int> >(rows,cols);
  };
  void calMetrics(dim3 grid,dim3 block) 
  {
    kernel2<<<grid, block>>>(d_data2->getDevData(), img->getDevData(), img->size());
    d_data2->show();
  };
  ~ConcreatePeri()
  {
    delete d_data2;
    d_data2 = NULL;
  }
};

template <class T>
void callTemplateMethod(T *proc,int rows,int cols,dim3 grid,dim3 block)
{
  if (proc!=NULL)
  {
    proc->templateMethod(rows,cols,grid,block);
    delete proc;
    proc = NULL;
  }
}




int main(int argc, char const *argv[])
{
  int rows = 10;
  int cols = 2;
	int arraySize = rows*cols;
	dim3 grid(1);
	dim3 block(arraySize);

  //下面这两个派生类的对象分别拥有一个基类对象，对象的地址是不同的，所以如果需要共用基类对象，需要将操作写到一个类中。
	ConcreatePeri *procPeri = new ConcreatePeri(rows,cols);
  callTemplateMethod(procPeri,rows,cols,grid,block);
  cout << "******************************************" << endl;
  ConcreateArea *procArea = new ConcreateArea(rows,cols);
  callTemplateMethod(procArea,rows,cols,grid,block);
  
	return 0;
}